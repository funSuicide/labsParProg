#include "hip/hip_runtime.h"
__global__ void matmul1(float* a, float* b, int n, float* c)
{
    float sum = 0.0f;

    int ia = (blockDim.y * blockIdx.y + threadIdx.y) * n;
    int ib = blockDim.x * blockIdx.x + threadIdx.x;
    int ic = ia + ib;

    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k * n];

    c[ic] = sum;
}

#define kernel matmul1
#include "mainGPU.h"